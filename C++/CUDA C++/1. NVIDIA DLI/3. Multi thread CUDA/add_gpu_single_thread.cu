
#include <hip/hip_runtime.h>
#include<iostream>
#include<math.h>

// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
  int index = threadIdx.x;
  int stride = blockDim.x;
  for (int i = index; i < n; i += stride)
      y[i] = x[i] + y[i];
}


int main(void)
{
    // initializes an integer variable N with the value 1,048,576
    int N = 1 << 20; // 1M elements using  left bitwise shift operator
    float *x, *y;

    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&x, N * sizeof(float)); // allocates memory of size N * sizeof(float) in the unified memory and points x to it
    hipMallocManaged(&y, N * sizeof(float)); // allocates memory of size N * sizeof(float) in the unified memory and points y to it

    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // Run kernel on 1M elements on the CPU
    add<<<1, 256>>>(N, x, y);   // call add function in the GPU with 1 block and 256 thread

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    //check for errors (all values should be 3.0f)
    float maxError = 0.0f; // initialize maxError to 0.0f
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i] - 3.0f)); // fabs is absolute value function
    std::cout << "Max error: " << maxError << std::endl;

    // Free memory from unified memory
    hipFree(x);
    hipFree(y);

    return 0;
}

